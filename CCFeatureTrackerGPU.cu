#include "hip/hip_runtime.h"
#include "kernel.h"

#include "hip/hip_runtime.h"
#include ""
#include "math_functions.h"
#include "hip/device_functions.h"
#include "device_functions_decls.h"

#include <stdio.h>

#include "opencv2\gpu\device\common.hpp"

using namespace cv::gpu;

namespace CCFeatureTrackerGPU_K {


	__global__ void innerProcessKernel( long cols, long rows, unsigned char* frameIn, unsigned char*  frameOut, int step )
	{
		const int x = threadIdx.x + blockIdx.x * blockDim.x;
        const int y = threadIdx.y + blockIdx.y * blockDim.y;


		/*int offset = x + y * blockDim.x * gridDim.x;*/
				

		if (x < cols && y < rows) {

			int offset = (x * 3) + y * step;

			frameOut[ offset ] =  frameIn[ offset + 2];
			frameOut[ offset + 1 ] = frameIn[ offset + 1];
			frameOut[ offset + 2 ] = frameIn[offset];
		}

		__syncthreads();
		
	}

	

	void innerProcess( long cols, long rows, unsigned char* frameIn, unsigned char*  frameOut, int step  )
	{
		
		hipError_t cudaStatus;
		dim3 block(16, 16);

		
        dim3 grid;
        grid.x = divUp(cols, block.x);
        grid.y = divUp(rows, block.y);

		// Launch a kernel on the GPU with one thread for each element.
		innerProcessKernel<<< grid, block>>>(cols, rows, frameIn, frameOut, step);

	    // Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
    
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}


Error:
		return;
	}

}