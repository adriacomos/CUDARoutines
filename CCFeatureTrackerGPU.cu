#include "hip/hip_runtime.h"
#include "kernels.h"

#include "hip/hip_runtime.h"
#include ""
#include "math_functions.h"
#include "hip/device_functions.h"
#include "device_functions_decls.h"

#include <stdio.h>

#include "opencv2\gpu\device\common.hpp"

using namespace cv::gpu;

namespace CUDARoutines {

namespace CCFeatureTrackerGPU_K {


	__global__ void innerProcessKernel( long cols, long rows, unsigned char* frameIn, unsigned char*  frameOut, int step )
	{
	
		const int x = threadIdx.x + blockIdx.x * blockDim.x;
        const int y = threadIdx.y + blockIdx.y * blockDim.y;

		// TODO		

		if (x < cols && y < rows) {
			int offset = (x * 3) + y * step;

			frameOut[ offset ] =  frameIn[ offset];
			frameOut[ offset + 1 ] = frameIn[ offset + 1];
			frameOut[ offset + 2 ] = frameIn[offset + 2];
		}

		__syncthreads();
		
	}

	

	void innerProcess( long cols, long rows, unsigned char* frameIn, unsigned char*  frameOut, int step  )
	{
		
		hipError_t cudaStatus;
		dim3 block(16, 16);

		
        dim3 grid;
        grid.x = divUp(cols, block.x);
        grid.y = divUp(rows, block.y);

		// Launch a kernel on the GPU with one thread for each element.
		innerProcessKernel<<< grid, block>>>(cols, rows, frameIn, frameOut, step);

	    // Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "innerProcessKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
    
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching innerProcessKernel!\n", cudaStatus);
			goto Error;
		}


Error:
		return;
	}

}

}